#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>
#include <math_functions.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <stdlib.h>
#include <time.h>
#include "RayTracer.h"

#include <stdio.h>

typedef unsigned int uint;

//size_t pitch;
__constant__ uint d_width;
__constant__ uint d_height;
__constant__ float3x4 d_invViewMatrix;
__constant__ float4 d_ambientLight;
__constant__ float4 d_backgroundColor;
__constant__ uint d_numLights;
__constant__ uint d_numTriangles;
__constant__ uint d_numSpheres;
__device__ Light *d_lights;
__device__ Triangle *d_triangles;
__device__ Sphere *d_spheres;

__device__ bool operator ==(float3 a, float3 b) { return a.x == b.x && a.y == b.y && a.z == b.z; }
__device__ bool operator !=(float3 a, float3 b) { return a.x != b.x && a.y != b.y && a.z != b.z; }
__device__ float3 operator +(float3 a, float3 b) { return make_float3(a.x + b.x, a.y + b.y, a.z + b.z); }
__device__ float3 operator -(float3 a, float3 b) { return make_float3(a.x - b.x, a.y - b.y, a.z - b.z); }
__device__ float3 operator *(float3 a, float3 b) { return make_float3(a.x * b.x, a.y * b.y, a.z * b.z); }
__device__ float3 operator -(float3 a, float s) { return make_float3(a.x - s, a.y - s, a.z - s); }
__device__ float3 operator -(float3 a) { return make_float3(-a.x , -a.y, -a.z); }
__device__ float3 operator *(float3 a, float s) { return make_float3(a.x * s, a.y * s, a.z * s); }
__device__ float3 operator /(float3 a, float s) { return make_float3(a.x / s, a.y / s, a.z / s); }
__device__ float Dot(float3 a, float3 b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
__device__ float Dot(float3 a, float4 b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
__device__ float Dot(float4 a, float4 b) { return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w; }
__device__ float Distance(float3 a, float3 b) { int dX, dY, dZ; dX = b.x - a.x; dY = b.y - a.y; dZ = b.z - a.z; return sqrtf(dX * dX + dY * dY + dZ * dZ); }
__device__ float3 Reflect(float3 v, float3 n) { return v - n * Dot(v, n) * 2; }
__device__ float3 Normalize(float3 v) { float xx, yy, zz, d; xx = v.x * v.x; yy = v.y * v.y; zz = v.z * v.z; d = sqrt(xx + yy + zz); return make_float3( v.x / d, v.y / d, v.z / d); }

__device__ bool operator ==(float4 a, float4 b) { return a.x == b.x && a.y == b.y && a.z == b.z && a.w == b.w; }
__device__ bool operator !=(float4 a, float4 b) { return a.x != b.x && a.y != b.y && a.z != b.z && a.w != b.w; }
__device__ float4 operator +(float4 a, float4 b) { return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w); }
__device__ float4 operator -(float4 a, float4 b) { return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w); }
__device__ float4 operator *(float4 a, float4 b) { return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w); }
__device__ float4 operator *(float4 a, float s) { return make_float4(a.x * s, a.y * s, a.z * s, a.w * s); }

__device__ float3 operator *(const float3x4 M, const float3 v) {
    float3 r;
    r.x = Dot(v, M.m[0]);
    r.y = Dot(v, M.m[1]);
    r.z = Dot(v, M.m[2]);
    return r;
}

__device__ float4 operator *(const float3x4 M, const float4 v) {
    float4 r;
    r.x = Dot(v, M.m[0]);
    r.y = Dot(v, M.m[1]);
    r.z = Dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}
__device__ bool operator ==(Sphere s1, Sphere s2) { return s1.r == s2.r && s1.p == s2.p; }
__device__ bool operator !=(Sphere s1, Sphere s2) { return s1.r != s2.r && s1.p != s2.p; }

__device__ bool operator ==(Triangle t1, Triangle t2) { return t1.v1 == t2.v1 && t1.v2 == t2.v2 && t1.v3 == t2.v3 && t1.n == t2.n; }
__device__ bool operator !=(Triangle t1, Triangle t2) { return t1.v1 != t2.v1 && t1.v2 != t2.v2 && t1.v3 != t2.v3 && t1.n != t2.n;  }

__device__ bool operator ==(Material m1, Material m2) {
                return m1.ambientColor == m2.ambientColor &&
                        m1.diffuseColor == m2.diffuseColor &&
                        m1.specularColor == m2.specularColor &&
                        m1.kR == m2.kR && m1.kT == m2.kT && m1.n == m2.n &&
                        m1.ambientStrength == m2.ambientStrength &&
                        m1.diffuseStrength == m2.diffuseStrength &&
                        m1.specularStrength == m2.specularStrength; }
        bool operator !=(Material m1, Material m2) {
                return m1.ambientColor != m2.ambientColor &&
                        m1.diffuseColor != m2.diffuseColor &&
                        m1.specularColor != m2.specularColor &&
                        m1.kR != m2.kR && m1.kT != m2.kT && m1.n != m2.n &&
                        m1.ambientStrength != m2.ambientStrength &&
                        m1.diffuseStrength != m2.diffuseStrength &&
			m1.specularStrength != m2.specularStrength; }
// Kernel functions
__global__ void trace(uint **d_pixelData);
__device__ float4 illuminate(Ray ray, int depth);
__device__ float intersects(Sphere *s, Ray r);
__device__ float intersects(Triangle *t, Ray r);
__device__ void *getClosestIntersection(Ray r, float3 *intersectPoint, ObjectType *type);
__device__ float4 calculateAmbient(Material *m);

// Copies the view matrix to device memory
void SetViewMatrix(float invViewMatrix[12])
{
    hipMemcpy(&d_invViewMatrix, invViewMatrix, sizeof(float) * 12, hipMemcpyHostToDevice);
}

// Copy scene data to device
void SetSceneData(float width, float height, float4 ambientLight, float4 backgroundColor,
		uint numLights, Light *lights,
		uint numTriangles, Triangle *triangles,
		uint numSpheres, Sphere *spheres)
{
	hipMemcpy(&d_width, &width, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(&d_height, &height, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(&d_ambientLight, &ambientLight, sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(&d_backgroundColor, &backgroundColor, sizeof(float4), hipMemcpyHostToDevice);

	//hipMallocPitch((void **)&d_pixelData, &pitch, d_width * sizeof(float4), d_height);
	//hipMalloc((void **)&d_pixelData, d_width * d_height * sizeof(float4));

	size_t sizeLights = numLights * sizeof(Light);
	hipMalloc((void **)&d_lights, sizeLights);
	hipMemcpy(&d_lights, lights, sizeLights, hipMemcpyHostToDevice);
	
	hipMemcpy(&d_numTriangles, &numTriangles, sizeof(uint), hipMemcpyHostToDevice);
	size_t sizeTriangles = numTriangles * sizeof(Triangle);
	hipMalloc((void **)&d_triangles, sizeTriangles);
	hipMemcpy(&d_triangles, triangles, sizeTriangles, hipMemcpyHostToDevice);
	
	hipMemcpy(&d_numSpheres, &numSpheres, sizeof(uint), hipMemcpyHostToDevice);
	size_t sizeSpheres = numSpheres * sizeof(Sphere);
	hipMalloc((void **)&d_spheres, sizeSpheres);
	hipMemcpy(&d_spheres, spheres, sizeSpheres, hipMemcpyHostToDevice);
}

void FreeSceneData()
{
	hipFree(d_lights);
	hipFree(d_triangles);
	hipFree(d_spheres);
}

void GetPixelData(uint **d_pixelData, dim3 gridSize, dim3 blockSize) {
	
	trace<<<gridSize, blockSize>>>(d_pixelData);
	//hipMemcpy(&pixelData, d_pixelData, d_width * d_height * sizeof(float4), hipMemcpyDeviceToHost);
	//hipMemcpy(&pixelData, d_pixelData, sizeof(float4) * width * height, hipMemcpyDeviceToHost);
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__global__
void trace(uint **d_pixelData) {
	int x, y;
	x = blockIdx.x * blockDim.x + threadIdx.x;
	y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x >= d_width) || (y >= d_height)) return;

    float u = (x / (float) d_width)*2.0f-1.0f;
    float v = (y / (float) d_height)*2.0f-1.0f;

    float4 p = d_invViewMatrix * make_float4(0.0f, 0.0f, 0.0f, 1.0f);

    Ray ray;
    ray.Position.x = p.x;
    ray.Position.y = p.y;
    ray.Position.z = p.z;
    ray.Direction = Normalize(make_float3(u, v, -2.0f));
    ray.Direction = d_invViewMatrix * ray.Direction;

	d_pixelData[x][y] = rgbaFloatToInt(illuminate(ray, 1));
}

__device__
float intersects(Triangle *t, Ray r) {
	//http://www.siggraph.org/education/materials/HyperGraph/raytrace/raypolygon_intersection.htm
	float3 n = t->n;
	float d = Dot(r.Direction, n);
	if(d == 0)
		return -1;
	return Dot((r.Position - t->v1), n) / d;
}

__device__
float intersects(Sphere *s, Ray ray) {
	// quadratic equation: t = (-b +/= sqrt(b * b - 4 * a * c)) / 2 * a

	// float a = Dot(ray.Direction, ray.Direction);
	// since ray direction is normalized, this will always = (1 += round error)
	// omitting a will save on calculations and reduce error

	float r = s->r;
	float3 p = s->p;
	float3 diff = ray.Position - p;
	float b = 2.0 * Dot(ray.Direction, diff);
	float c = Dot(diff, diff) - r * r;

	// approximate if below precision quantum
	if (c < .001)//std::numeric_limits<float>::epsilon())
        return 0;

	float d = b * b - 4.0 * c;
	// unreal, no root
	if(d < 0)
		return -1;

	float e = sqrt(d);

	// first root
	float t1 = (-b - e) / 2.0;
	if(t1 >= 0)
		return t1;

	// second root
	float t2 = (-b + e) / 2.0;
	if(t2 >= 0)
		return t2;

	return -1;
}


// kernel and device functions

/// <summary>
/// Finds the closest intersected RTObjectand sets the intersectPoint float3.
/// </summary>
/// <param name="ray">The ray to test RTObjectintersections.</param>
/// <param name="intersectPoint">The float3 to hold the intersection data.</param>
/// <returns>The closest intersected RTObject, or null if no RTObject is intersected.</returns>
__device__
void *getClosestIntersection(Ray ray, float3 *intersectPoint, ObjectType *type)
{
	float minDist = HIP_INF_F;
	float curDist;
	void *intersected = NULL;

	uint i;
	for(i = 0; i < d_numTriangles; ++i)
	{
		Triangle *t = &d_triangles[i];
		curDist = intersects(t, ray);
		if (curDist > 0 && curDist < minDist)
		{
			minDist = curDist;
			intersected = (void *)t;
			*type = T_Triangle;
		}
	}

	for(i = 0; i < d_numSpheres; i++)
	{
		Sphere *s = &d_spheres[i];
		curDist = intersects(s, ray);
		if (curDist > 0 && curDist < minDist)
		{
			minDist = curDist;
			intersected = (void *)s;
			*type = T_Sphere;
		}
	}

	if(intersected)
		*intersectPoint = ray.Position + ray.Direction * minDist;

	return intersected;
}

__device__
float4 calculateAmbient(Material *m)
{
	float4 ambientLight = d_ambientLight;
	if(m) ambientLight = ambientLight * m->ambientColor * m->ambientStrength;
	return ambientLight;
}

__device__
float4 calculateDiffuse(Material *m, float3 worldCoords, Light l, float3 normal, float3 lightVector) {
	float4 diffuseLight = l.Color;
	if (m)
		diffuseLight = diffuseLight *
			fabs(Dot(lightVector, normal)) * 
			m->diffuseColor * 
			m->diffuseStrength;
	return diffuseLight;
}

__device__
float4 calculateSpecular(Material *m, float3 worldCoords, Light l, float3 normal, float3 lightVector, float3 viewVector) {
	float3 reflectedVector = Reflect(lightVector, normal);
	float dot = Dot(reflectedVector, viewVector);

	if (dot >= 0)
	    return make_float4(0, 0, 0, 0);

	float4 specularLight = l.Color;
	
	if (m)
	{
		specularLight = specularLight *
			fabs(Dot(lightVector, normal) * pow(dot, m->exponent)) *
			m->specularColor *
			m->specularStrength;
	}

	return specularLight;
}

/// <summary>
/// Spawns a shadow ray.
/// </summary>
/// <param name="intersectPoint">Origin of the ray</param>
/// <param name="intersectedObject">World object that was intersected</param>
/// <param name="intersectNormal">Normal of the world object at the intersection point</param>
/// <param name="viewVector">Camera view vector.</param>
/// <param name="depth">current recursion depth.</param>
/// <returns></returns>
__device__
float4 spawnShadowRay(float3 intersectPoint, void *intersectedObject, Material *m, /*ObjectType t,*/ float3 intersectNormal, float3 viewVector, int depth)
{
	float4 diffuseTotal;
	float4 specularTotal;

	uint i;
	for(i = 0; i < d_numLights; ++i)
	{
		Light light = d_lights[i];

		// Spawn a shadow ray from the intersection point to the light source
		float3 lightVector = Normalize(light.Position - intersectPoint);

		// but only if the intersection is facing the light source
		float facing = Dot(intersectNormal, lightVector);
		if (facing < 0)
		{
			/*Ray shadowRay;
			shadowRay.Position = intersectPoint;
			shadowRay.Direction = lightVector;

			// Check if the shadow ray reaches the light before hitting any other object
			float dist = Distance(intersectPoint, light.Position);
			bool shadowed = false;

			float4 shadowLight;

			uint k;
			for(k = 0; k < numTriangles; ++k)
			{
				Triangle t* = &d_triangles[k];
				if (*t != intersectedObject)
				{
					float curDist = rt->Intersects(shadowRay);
					if (curDist > 0 && curDist < dist)
					{
						dist = curDist;
						shadowed = true;

#if !TRANSMIT_SHADOW
						break;
#else
						Material *m = rt->GetMaterial();
						if (m->kT > 0)
						{
							float3 incidentVector = (intersectPoint - shadowRay.Position).Normalize();
							float3 shadowIntersect = shadowRay.Position + (shadowRay.Direction * curDist);
							float3 shadowNormal = rt->GetIntersectNormal(shadowIntersect);

							shadowLight = (shadowLight + spawnTransmissionRay(depth, shadowIntersect, rt, shadowNormal, incidentVector)) * m->kT;
						}
						else
						{
							shadowLight.x = 0;
							shadowLight.y = 0;
							shadowLight.z = 0;
							shadowLight.w = 0;
							break;
						}
#endif
					}
				}
			}*/

			/*if (shadowed)
			{
				diffuseTotal = diffuseTotal + intersectedObject->calculateDiffuse(intersectPoint, intersectNormal, light, lightVector) * shadowLight;
				specularTotal = specularTotal + intersectedObject->calculateSpecular(intersectPoint, intersectNormal, light, lightVector, viewVector) * shadowLight;
			}
			else
			{*/
				diffuseTotal = diffuseTotal + calculateDiffuse(m, intersectPoint, light, intersectNormal, lightVector);
				specularTotal = specularTotal + calculateSpecular(m, intersectPoint, light, intersectNormal, lightVector, viewVector);
			//}

		}
	}

	return diffuseTotal * m->diffuseStrength + specularTotal * m->specularStrength;
}

__device__
float4 illuminate(Ray ray, int depth) {
    float3 intersectPoint;
    ObjectType type;
    void *rt = getClosestIntersection(ray, &intersectPoint, &type);

    if (rt)
    {
        float3 intersectNormal;
		Material *m;
		if(type == T_Sphere)
		{
			Sphere *s = (Sphere *)rt;
			intersectNormal = Normalize(intersectPoint - s->p);
			m = &(s->m);
		} else {
			Triangle *t = (Triangle *)rt;
			intersectNormal = t->n;
			m = &(t->m);
		}

        //float3 viewVector = Normalize(ray.Position - intersectPoint);
        float3 viewVector = -ray.Direction;
        float4 totalLight = calculateAmbient(m);
        totalLight = totalLight + spawnShadowRay(intersectPoint, rt, m, intersectNormal, viewVector, depth);

        /*if (depth < recursionDepth)
        {
            float3 incidentVector = Normalize(intersectPoint - ray.Position);

            // Material is reflective
            if (m->kR > 0)
            {
                float3 dir = incidentVector.Reflect(intersectNormal);
                Ray reflectionRay;
                reflectionRay.Position = intersectPoint;
                reflectionRay.Direction = dir;
                totalLight = totalLight + (illuminate(reflectionRay, depth + 1) * m->kR);
            }

            // Material is transparent
            if (m->kT > 0)
            {
                totalLight = totalLight + spawnTransmissionRay(depth, intersectPoint, rt, intersectNormal, incidentVector);
            }
        }*/

        return totalLight;
    }
    else
    {
        return d_backgroundColor;
    }
}
/*
/// <summary>
/// Spawns a recursive, transmitted (refracted) ray.
/// </summary>
/// <param name="depth">Current recursion depth</param>
/// <param name="intersectPoint">Origin of the ray</param>
/// <param name="intersectedObject">World object that was intersected</param>
/// <param name="intersectNormal">Normal of the world object at the intersection point</param>
/// <param name="totalLight">Total light to contribute to.</param>
/// <param name="incidentVector">Ray direction incident to intersection.</param>
__device__
float4 spawnTransmissionRay(int depth, float3 intersectPoint, RTObject *intersectedObject, float3 intersectNormal, float3 incidentVector)
{
	float n;

	// Parity check
	Material *m = intersectedObject->GetMaterial();
	if (depth % 2 == 0)
	{
		// assuming outside to inside
		n = m->n;
	}
	else
	{
		// assuming inside to outside
		n = 1 / m->n;
		intersectNormal = intersectNormal * -1;
	}

	float dot = incidentVector.Dot(intersectNormal);
	float discriminant = 1 + ((n * n) * ((dot * dot) - 1));

	if (discriminant < 0)
	{
		// simulate total internal reflection
		float3 dir = incidentVector.Reflect(intersectNormal);
		Ray reflectionRay;
		reflectionRay.Position = intersectPoint;
		reflectionRay.Direction = dir;
		return illuminate(reflectionRay, depth + 1) * m->n;
	}
	else
	{
		float3 dir = incidentVector * n + (intersectNormal * (n * dot - sqrt(discriminant)));
		Ray transRay;
		transRay.Position = intersectPoint;
		transRay.Direction = dir;
		return illuminate(transRay, depth + 1) * intersectedObject->GetMaterial()->kT;
	}
}
*/
